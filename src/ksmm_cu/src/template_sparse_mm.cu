#include "hip/hip_runtime.h"
// -*- c++ -*-

#include <cassert>
#include <ctime>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <getopt.h>
#include <omp.h>
#include <random>
#include <stdexcept>
#include <stdio.h>

#defineFLOAT4orHALF2

#ifdef FLOAT4
#include "kernels_float4.cuh"
#endif

#ifdef HALF2
#include "kernels_half2.cuh"
#endif

// using namespace nvcuda;

char name[200];
FILE *fout;

int MF(int row, int col, int ld, int mf);

int MF(int row, int col, int ld, int mf)
{
  // mf = 0 reads in row-major format
  // mf = 1 reads in column-major format
  return (mf == 0) ? row * ld + col : col * ld + row;
}

int main(int argc, char **argv)
{
  int nargs = 0, arg_i;
  char kernel_name[100];
  int deviceId = 0;
  int nrepeats = 100;
  const option long_opts[] = {{"kernel", required_argument, nullptr, 'n'},
                              {"device", required_argument, nullptr, 'd'},
			      {"nrepeats", required_argument, nullptr, 'r'}};
  while ((arg_i = getopt_long(argc, argv, "n:d:r:h", long_opts, nullptr)) != -1)
  {
    switch (arg_i)
    {
    case 'n':
      sprintf(kernel_name, "%s", optarg);
      nargs++;
      break;
    case 'd':
      deviceId = (int)atof(optarg);
      nargs++;
      break;
    case 'r':
      nrepeats = (int)atof(optarg);
      nargs++;
      break;
    case 'h':
      break;
    default:
      printf("\n");
      break;
    }
  }

  int seed = 1;
  std::mt19937_64 mt(seed);
  std::uniform_real_distribution<float> u01(.0, 1.);

  // Grid
  const int batch_size = xBATCHSIZEx;
  const int input_size = xINPUTSIZEx;
  const int output_size = xOUTPUTSIZEx;
  int dim1, dim2;
  dim3 blocksPerGrid(1, 1, 1);
  dim3 threadsPerBlock(1, 1, 1);
  // Grid for the tensor cores kernel
  const int WMMA_Y = xWMMA_Yx;
  const int WMMA_X = xWMMA_Xx;
  const int WMMA_K = xWMMA_Kx;
  int nwarpsY = xNWARPSYx;
  int nwarpsX = xNWARPSXx;
  // Tile dimensions
  const int TILEX = xTILEXx;
  const int TILEK = xTILEKx;
  const int TILEY = xTILEYx;
  const int TX = xTXx;
  const int TY = xTYx;

  // Allocate matrices in host memory
  float *input;
  float *valuesT;
  float *bfactor;
  // half precision
  half *h_input;
  half *h_valuesT;
  half *h_bfactor;
  // To store the output of matrix multiplication
  float *gpu_output, *true_output;
  half *h_gpu_output;

  // Allocate matrices in device memory
  float *d_input;
  float *d_bfactor;
  float *d_valuesT;
  float *d_output;
  // half precision
  half *d_h_input;
  half *d_h_bfactor;
  half *d_h_valuesT;
  half *d_h_output;

  // alpha and beta for cuBlas routines
  const float alpha = 1.0;
  const float beta = 0.0;
  const half h_alpha = 1.0;
  const half h_beta = 0.0;

  // Leading dimensions
  int lda, ldb, ldc, mf;

  // Check if user asks for cuBlas routines
  bool is_cublas = false;
  if (strcmp(kernel_name, "cublas_factor0_fp16") == 0 ||
      strcmp(kernel_name, "cublas_stride_factor0_fp16") == 0 ||
      strcmp(kernel_name, "cublas_factor0_fp32") == 0 ||
      strcmp(kernel_name, "cublas_stride_factor0_fp32") == 0)
    is_cublas = true;

  // Check if half-precision
  bool hp = false;
  if (strcmp(kernel_name, "cublas_factor0_fp16") == 0 ||
      strcmp(kernel_name, "cublas_stride_factor0_fp16") == 0 ||
      strcmp(kernel_name, "kernel_bs_first_half2") == 0 ||
      strcmp(kernel_name, "kernel_bs_last_half2") == 0)
    hp = true;

  // Check if batch-size last position kernel
  bool bs_last = false;
  if (strcmp(kernel_name, "kernel_bs_last_float4") == 0 ||
      strcmp(kernel_name, "kernel_bs_last_half2") == 0)
    bs_last = true;

  bool check_output = false;
  bool take_too_long = false;
  bool debug = false;

  // CUDA device properties
  int nDevices;
  int maxDynamicSharedMem;
  int maxSharedMem;
  int maxThreadsPerBlock;
  int maxBlockDimX, maxBlockDimY;
  int maxGridDimX, maxGridDimY;
  int maxRegistersPerBlock;
  hipDataType cuda_data_type = hp ? HIP_R_16F : HIP_R_32F;
  hipEvent_t cstart, cend;
  float ct, ct_cutoff = 1e6, meant, stdt, *ts = new float[nrepeats]();

  // Mean-Square-Error between CPU and GPU computation
  float mse = 0.0;

  // Grid size as a function of Butterfly factor
  // Dimensions (cf overleaf)
  dim1 = input_size;
  dim2 = output_size;
  printf("batch=%i input_size=%i output_size=%i\n", batch_size, input_size,
         output_size);

  threadsPerBlock.x = xNTHREADS_FLOAT4x;
  threadsPerBlock.y = 1;
  blocksPerGrid.x = (dim2 + TILEX - 1) / TILEX;
  blocksPerGrid.y = (batch_size + TILEY - 1) / TILEY;

  if (!is_cublas)
  {
    printf("TILEX=%i TILEK=%i TILEY=%i TX=%i TY=%i\n", TILEX, TILEK, TILEY, TX, TY);
    printf("threadsPerBlock=%i %i\n", threadsPerBlock.x, threadsPerBlock.y);
    printf("blocksPerGrid=%i %i\n", blocksPerGrid.x, blocksPerGrid.y);
  }

  // Shared memory
  int smem = (hp ? sizeof(half) : sizeof(float)) *
    ((TILEY * TILEK + TILEK * TILEX) + TILEY * TILEX);

  if (strcmp(kernel_name, "cublas_stride_factor0_fp16") == 0 ||
      strcmp(kernel_name, "cublas_stride_factor0_fp32") == 0)
  {
    // If CuBlas stride tranpose (row to column major format)
    // Move everything from row to column major format ?
    lda = batch_size;
    ldb = dim1;
    ldc = batch_size;
    mf = 1;
  }
  else
  {
    lda = dim1;
    ldb = dim2;
    ldc = dim2;
    mf = 0;
  }

  // Host device allocations ...
  printf("Host device matrices allocations ...\n");
  // Random matrix
  printf("Random matrix ...\n");
  input = new float[hp ? 1 : batch_size * dim1]();
  h_input = new half[hp ? batch_size * dim1 : 1]();
  int istride = debug ? 1 : 200;
  int nthreads = 16;
  int bpert = (int)std::ceil(batch_size / (double)nthreads);
  bpert = istride * ((int)(bpert / istride) + 1);
  int row, col;
#pragma omp parallel for num_threads(nthreads)
  for (int n = 0; n < nthreads; n++) {
    for (int b = n * bpert; b < min(batch_size, (n + 1) * bpert); b += istride) {
      for (int i = 0; i < dim1; i++) {
	row = (int)(batch_size * u01(mt));
	col = i;
	if (!debug) {
	  if (hp)
	    h_input[(bs_last) ? MF(row, col, batch_size, 1)
		    : MF(row, col, lda, mf)] =
	      __float2half(u01(mt));
	  else
	    input[(bs_last) ? MF(row, col, batch_size, 1)
		  : MF(row, col, lda, mf)] = u01(mt);
	} else {
	  if (hp)
	    h_input[(bs_last) ?
		    MF(row, col, batch_size, 1) : MF(row, col, lda, mf)] = __float2half(1.0f);
	  else
	    input[(bs_last) ?
		  MF(row, col, batch_size, 1) : MF(row, col, lda, mf)] = (row == 0) ? 1.0 : 0.0;
	}
      }
    }
  }

  // Butterfly factor
  // tuple (a, b, c, d)
  // B = kron(Id_{a,a}, kron(1_{b,c}, Id_{d,d}))
  // There is 'a' super-blocks of shape (b * d, c * d).
  // Number of non-zero per super-block is
  // b per column and c per row.
  // We would like to compute X @ B^T.
  // X shape is (batch, a * c * d).
  // B^T shape is (dim1, dim2).
  // dim1 = a * c * d
  // dim2 = a * b * d
  printf("Butterfly factor (%i,%i,%i,%i) ...\n", xax, xbx, xcx, xdx);
  int NNZ = xax * xbx * xcx * xdx;
  valuesT = new float[hp ? 1 : NNZ]();
  h_valuesT = new half[hp ? NNZ : 1]();
  bfactor = new float[is_cublas ? (hp ? 1 : dim1 * dim2) : 1]();
  h_bfactor = new half[is_cublas ? (hp ? dim1 * dim2 : 1) : 1]();
  int count = 0;
  float value;
  half h_value;
  // Loop over the super-block
  int ii, jj;
  for (int aa = 0; aa < xax; aa++)
    {
      // First row and first column of the current super-block
      ii = aa * dim1 / xax;
      jj = aa * dim2 / xax;
      // Loop over the columns inside super-block and
      // store the index of rows of non-zero entries
      for (int m = 0; m < xdx; m++)
	{
	  for (int j = m; j < (dim2 / xax); j += xdx)
	    {
	      for (int i = m; i < (dim1 / xax); i += xdx)
		{
		  // inside super-block
		  if (hp)
		    {
		      // handle half case
		      h_value = debug ? __float2half(1.0f) : __float2half(u01(mt));
		      // There is b * c * d non-zero per super-block
		      // Current super-block is aa
		      h_valuesT[aa * xbx * xcx * xdx + m * xbx * xcx + (i / xdx) * (dim2 / (xax * xdx)) + j / xdx] = h_value;
		      if (is_cublas)
			h_bfactor[MF(ii + i, jj + j, ldb, mf)] = h_value;
		    }
		  else
		    {
		      value = debug ? 1.0 : u01(mt);
		      // There is b * c * d non-zero per super-block
		      // Current super-block is aa
		      valuesT[aa * xbx * xcx * xdx + m * xbx * xcx + (i / xdx) * (dim2 / (xax * xdx)) + j / xdx] = value;
		      if (is_cublas)
			bfactor[MF(ii + i, jj + j, ldb, mf)] = value;
		    }
		}
	    }
	}
    }

  // output
  gpu_output = new float[hp ? 1 : batch_size * dim2]();
  h_gpu_output = new half[hp ? batch_size * dim2 : 1]();
  true_output = new float[batch_size * dim2]();

  // Loop over devices
  printf("From host to device ...\n");
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++)
  {
    if (i != deviceId)
      continue;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    hipDeviceGetAttribute(&maxDynamicSharedMem,
                           hipDeviceAttributeSharedMemPerBlockOptin, i);
    hipDeviceGetAttribute(&maxSharedMem, hipDeviceAttributeMaxSharedMemoryPerBlock,
                           i);
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock,
                           i);
    hipDeviceGetAttribute(&maxBlockDimX, hipDeviceAttributeMaxBlockDimX, i);
    hipDeviceGetAttribute(&maxBlockDimY, hipDeviceAttributeMaxBlockDimY, i);
    hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX, i);
    hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY, i);
    hipDeviceGetAttribute(&maxRegistersPerBlock,
                           hipDeviceAttributeMaxRegistersPerBlock, i);
    printf("device %i/%i\n", i, nDevices);
    printf("maxBlockDimX=%i maxBlockDimY=%i maxGridDimX=%i "
           "maxGridDimY=%i MaxThreadsPerBlock=%i\n",
           maxBlockDimX, maxBlockDimY, maxGridDimX, maxGridDimY,
           maxThreadsPerBlock);
    printf("registers per block=%i\n", maxRegistersPerBlock);
    printf("smem=%i/%i\n", smem, maxSharedMem);
  }
  assert(xNTHREADS_FLOAT4x <= maxThreadsPerBlock);
  assert(xNTHREADS_HALF2x <= maxThreadsPerBlock);
  assert(smem < maxSharedMem);
  assert(blocksPerGrid.x <= maxGridDimX && blocksPerGrid.y <= maxGridDimY);
  assert(threadsPerBlock.x <= maxBlockDimX &&
         threadsPerBlock.y <= maxBlockDimY);

  // Set device
  hipSetDevice(deviceId);

  // Allocate matrices in device memory
  printf("device memory allocation ...\n");
  if (hp)
  {
    hipMalloc(&d_h_input, sizeof(half) * batch_size * dim1);
    if (is_cublas)
      hipMalloc(&d_h_bfactor, sizeof(half) * dim1 * dim2);
    hipMalloc(&d_h_valuesT, sizeof(half) * NNZ);
    hipMalloc(&d_h_output, sizeof(half) * batch_size * dim2);
  }
  else
  {
    hipMalloc(&d_input, sizeof(float) * batch_size * dim1);
    if (is_cublas)
      hipMalloc(&d_bfactor, sizeof(float) * dim1 * dim2);
    hipMalloc(&d_valuesT, sizeof(float) * NNZ);
    hipMalloc(&d_output, sizeof(float) * batch_size * dim2);
  }

  // Copy data from host memory to device memory
  printf("host to device memory ...\n");
  if (hp)
  {
    hipMemcpy(d_h_input, h_input, sizeof(half) * batch_size * dim1,
               hipMemcpyHostToDevice);
    if (is_cublas)
      hipMemcpy(d_h_bfactor, h_bfactor, sizeof(half) * dim1 * dim2,
                 hipMemcpyHostToDevice);
    hipMemcpy(d_h_valuesT, h_valuesT, sizeof(half) * NNZ,
               hipMemcpyHostToDevice);
  }
  else
  {
    hipMemcpy(d_input, input, sizeof(float) * batch_size * dim1,
               hipMemcpyHostToDevice);
    if (is_cublas)
      hipMemcpy(d_bfactor, bfactor, sizeof(float) * dim1 * dim2,
                 hipMemcpyHostToDevice);
    hipMemcpy(d_valuesT, valuesT, sizeof(float) * NNZ, hipMemcpyHostToDevice);
  }

  // Multiple runs of the kernel
  // ikernel is 0 if no valid kernels found
  int ikernel =
      1 * (int)(strcmp(kernel_name, "cublas_stride_factor0_fp16") == 0 ||
                strcmp(kernel_name, "cublas_stride_factor0_fp32") == 0) +
      2 * (int)(strcmp(kernel_name, "cublas_factor0_fp16") == 0 ||
                strcmp(kernel_name, "cublas_factor0_fp32") == 0) +
      3 * (int)(strcmp(kernel_name, "kernel_bs_first_float4") == 0 ||
                strcmp(kernel_name, "kernel_bs_first_half2") == 0 ||
                strcmp(kernel_name, "kernel_bs_last_float4") == 0 ||
                strcmp(kernel_name, "kernel_bs_last_half2") == 0);

  // Set CUDA time threshold to the minimum CUDA time ???
  int i0, i1, i2, i3, i4, i6, i7, i8, i9, i10, i11, i12;
  int s0, s1, s2, s3, s4, s6, s7, s8, s9, s10;
  s0 = s1 = s2 = s3 = s4 = s6 = s7 = s8 = s9 = s10 = -1;
  float f0, f1; //, f2;
  float tmin = 1e6;
  int nhp = 0;
  count = 0;
  sprintf(name, "%s.out", kernel_name);
  fout = fopen(name, "r");
  if (fout != NULL)
    {
      while (!feof(fout))
	{
	  if (count > 0)
	    {
	      fscanf(
		     fout,
		     "%i %i %i %i %i %i %i %i %i %i %i %i %f %f %*e\n",
		     &i0, &i1, &i2, &i3, &i4, &i6, &i7, &i8, &i9, &i10, &i11,
		     &i12, &f0, &f1);
	      if (xBATCHSIZEx == i0 && xax == i1 && xbx == i2 && xcx == i3 &&
		  xdx == i4)
		{
		  s0 = i0;
		  s1 = i1;
		  s2 = i2;
		  s3 = i3;
		  s4 = i4;
		  s6 = i6;
		  s7 = i7;
		  s8 = i8;
		  s9 = i9;
		  s10 = i10;
		  tmin = fmin(tmin, f0);
		  nhp++;
		}
	    }
	  else
	    fscanf(fout, "%*[^\n]\n");
	  count++;
	}
      fclose(fout);
    }
  if (count > 1 && s0 != -1)
    {
      ct_cutoff = 1.25 * tmin;
      printf("tmin=%f (%i, %i %i %i %i, %i %i %i %i %i)\n", tmin, s0, s1, s2,
             s3, s4, s6, s7, s8, s9, s10);
    }
  if(0 && nhp > 100)
    return 0;
  // ???

  const std::clock_t c_start = std::clock();
  hipblasHandle_t cublasHandle;
  hipblasStatus_t hipblasStatus_t;
  float cum_time = 0.0;
  long long int stride_input = xcx * batch_size;
  long long int stride_values = xbx * xcx;
  long long int stride_output = xbx * batch_size;
  int batch_count = dim1 / xbx;
  // GPU warmup: w = 0
  printf("Warmup and then repeat %i runs ...\n", nrepeats);
  take_too_long = false;
  for (int w = 0; w < 2; w++)
  {
    cum_time = 0.0;
    for (int i = 0; i < ((w == 0) ? 100000000 : nrepeats); i++)
    {
      hipEventCreate(&cstart);
      hipEventRecord(cstart, 0);
      switch (ikernel)
      {
      case 0:
        throw std::invalid_argument("Did not find the kernel.");
        break;
      case 1:
        // CuBlas uses column-major format
        // However, we create matrices A and B using row-major format
        // A^T and B^T are in column-major format
        // Therefore, cublas computes C^T = (A @ B)^T that is in column-major
        // format
        if (w == 0 && i == 0)
          hipblasCreate(&cublasHandle);
        // CUBLAS_GEMM_ALGO0_TENSOR_OP to CUBLAS_GEMM_ALGO15_TENSOR_OP
        // CUBLAS_GEMM_ALGO0 to CUBLAS_GEMM_ALGO23
        if (xdx == 1)
        {
          // Butterfly factor 0
          // |A B| @ |D1  0| = |A @ D1 B @ D2|
          // |C D|   |0  D2|   |C @ D1 D @ D2|
          // We consider the batch as an horizontal stack
          // of rectangular matrices batch_size x (b * d, c * d)
          // Use CSR format
          if (strcmp(kernel_name, "cublas_stride_factor0_fp16") == 0)
          {
            hipblasStatus_t = hipblasGemmStridedBatchedEx(
                cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, batch_size, xcx, xbx,
                &h_alpha, d_h_input, cuda_data_type, batch_size, stride_input,
                d_h_valuesT, cuda_data_type, xbx, stride_values, &h_beta,
                d_h_output, cuda_data_type, batch_size, stride_output,
                batch_count, cuda_data_type, xCUBLAS_GEMM_ALGO_TENSOR_OPx);
          }
          else
          {
            hipblasStatus_t = hipblasGemmStridedBatchedEx(
                cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, batch_size, xcx, xbx,
                &alpha, d_input, cuda_data_type, batch_size, stride_input,
                d_valuesT, cuda_data_type, xbx, stride_values, &beta, d_output,
                cuda_data_type, batch_size, stride_output, batch_count,
                cuda_data_type, xCUBLAS_GEMM_ALGOx);
          }
        }
        else
	  throw std::runtime_error("Not implemented for d=1.");
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
        {
          printf("%i %i %i %i %i %i\n", hipblasStatus_t, HIPBLAS_STATUS_SUCCESS,
                 HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED,
                 HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_EXECUTION_FAILED);
          throw std::invalid_argument("hipblasStatus_t != HIPBLAS_STATUS_SUCCESS");
        }
        if (w == 1 && i == (nrepeats - 1))
          hipblasDestroy(cublasHandle);
        break;
      case 2:
        // CuBlas uses column-major format
        // However, we create matrices A and B using row-major format
        // A^T and B^T are in column-major format
        // Therefore, cublas computes C^T = (A @ B)^T that is in column-major
        // format
        if (w == 0 && i == 0)
          hipblasCreate(&cublasHandle);
        // CUBLAS_GEMM_ALGO0_TENSOR_OP to CUBLAS_GEMM_ALGO15_TENSOR_OP
        if (strcmp(kernel_name, "cublas_factor0_fp16") == 0)
          hipblasStatus_t = hipblasGemmEx(
              cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim2, batch_size, dim1,
              &h_alpha, d_h_bfactor, cuda_data_type, dim2, d_h_input,
              cuda_data_type, dim1, &h_beta, d_h_output, cuda_data_type, dim2,
              cuda_data_type, xCUBLAS_GEMM_ALGO_TENSOR_OPx);
        else
          hipblasStatus_t = hipblasGemmEx(
              cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim2, batch_size, dim1,
              &alpha, d_bfactor, cuda_data_type, dim2, d_input, cuda_data_type,
              dim1, &beta, d_output, cuda_data_type, dim2, cuda_data_type,
              xCUBLAS_GEMM_ALGOx);
        if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
        {
          printf("%i %i %i %i %i %i\n", hipblasStatus_t, HIPBLAS_STATUS_SUCCESS,
                 HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED,
                 HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_EXECUTION_FAILED);
          throw std::invalid_argument("hipblasStatus_t != HIPBLAS_STATUS_SUCCESS");
        }
        if (w == 1 && i == (nrepeats - 1))
          hipblasDestroy(cublasHandle);
        break;
      case 3:
#ifdef FLOAT4
        if (strcmp(kernel_name, "kernel_bs_first_float4") == 0)
        {
          kernel_bs_first_float4<xTILEXx, xTILEKx, xTILEYx, xTXx, xTYx>
              <<<blocksPerGrid, threadsPerBlock>>>(d_input, d_valuesT,
						   batch_size, d_output,
						   xax, xbx, xcx, xdx);
        }
        if (strcmp(kernel_name, "kernel_bs_last_float4") == 0)
        {
          kernel_bs_last_float4<xTILEXx, xTILEKx, xTILEYx, xTXx, xTYx>
              <<<blocksPerGrid, threadsPerBlock>>>(d_input, d_valuesT,
						   batch_size, d_output,
						   xax, xbx, xcx, xdx);
        }
#endif
#ifdef HALF2
        if (strcmp(kernel_name, "kernel_bs_first_half2") == 0)
        {
          kernel_bs_first_half2<xTILEXx, xTILEKx, xTILEYx, xTXx, xTYx>
              <<<blocksPerGrid, threadsPerBlock>>>(d_h_input, d_h_valuesT,
                                                   batch_size, d_h_output,
                                                   xax, xbx, xcx, xdx);
        }
        if (strcmp(kernel_name, "kernel_bs_last_half2") == 0)
        {
          kernel_bs_last_half2<xTILEXx, xTILEKx, xTILEYx, xTXx, xTYx>
              <<<blocksPerGrid, threadsPerBlock>>>(d_h_input, d_h_valuesT,
                                                   batch_size, d_h_output,
                                                   xax, xbx, xcx, xdx);
        }
#endif
        break;
      default:
        throw std::invalid_argument("Did not find the kernel.");
        break;
      }
      hipEventCreate(&cend);
      hipEventRecord(cend, 0);
      hipEventSynchronize(cend);
      hipEventElapsedTime(&ct, cstart, cend);
      cum_time += ct;
      if (w == 1 && i == (nrepeats - 1))
        printf("End warmup and repeats.\n");
      // Do 100 ms warmup
      if (w == 0 && cum_time > 100.0)
        break;
      // Save time if not warmup
      if (w == 1)
        ts[i] = ct;
      // Does it take too much time ?
      if (w == 1 && ct > ct_cutoff)
      {
        take_too_long = true;
        for (int j = 0; j < nrepeats; j++)
          ts[j] = 3600.0 * 1e3;
        break;
      }
    }
    if (take_too_long)
      break;
  }
  if (take_too_long)
  {
    printf("Take too long (%f > %f), exit.\n", ct, ct_cutoff);
    return 0;
  }

  const std::clock_t c_end = std::clock();
  meant = 0.0f;
  for (int i = 0; i < nrepeats; i++)
    meant += ts[i];
  meant /= nrepeats;
  stdt = 0.0f;
  for (int i = 0; i < nrepeats; i++)
    stdt += pow(ts[i] - meant, 2.);
  stdt = sqrt(stdt / (nrepeats - 1));
  printf("factor cuda=%f +/- %f ms\n", meant, stdt);

  // Copy data from device memory to host memory
  if (hp)
    hipMemcpy(h_gpu_output, d_h_output, sizeof(half) * batch_size * dim2,
               hipMemcpyDeviceToHost);
  else
    hipMemcpy(gpu_output, d_output, sizeof(float) * batch_size * dim2,
               hipMemcpyDeviceToHost);

  check_output = (bool)(meant <= tmin);
  if (check_output)
  {
    // batch / Butterfly factor multiplication
    printf("use CPU to compute the output\n");
    const std::clock_t cpu_start = std::clock();
    if (hp)
      {
	// Butterfly factor
#pragma omp parallel for num_threads(nthreads)
	for (int n = 0; n < nthreads; n++)
	  {
	    for (int i = n * bpert; i < min(batch_size, (n + 1) * bpert);
		 i += istride)
	      {
		for (int m = 0; m < xdx; m++)
		  {
		    // TODO: comment about the index access
		    // We store all the columns j such that they have the same j % d
		    // in a contiguous way. The size of such group is dim2 // d.
		    for (int j = m; j < dim2; j += xdx)
		      {
			for (int k = m; k < dim1; k += xdx)
			  {
			    // Same super-block (a super-blocks (b * d, c * d)) ?
			    if ((j / (dim2 / xax)) != (k / (dim1 / xax)))
			      continue;
			    // There is b * c * d non-zero per super-block
			    true_output[MF(i, j, ldc, mf)] +=
			      __half2float(h_input[MF(i, k, (bs_last) ? batch_size : lda, (bs_last) ? 1 : mf)]) *
			      __half2float(h_valuesT[(k / (dim1 / xax)) * xbx * xcx * xdx +
						     m * xbx * xcx +
						     ((k - (dim1 / xax) * (k / (dim1 / xax))) / xdx) * (dim2 / (xax * xdx)) +
						     (j - (dim2 / xax) * (j / (dim2 / xax))) / xdx]);
			  }
		      }
		  }
	      }
	  }
      }
    else
      {
	// Butterfly factor
#pragma omp parallel for num_threads(nthreads)
	for (int n = 0; n < nthreads; n++)
	  {
	    for (int i = n * bpert; i < min(batch_size, (n + 1) * bpert);
		 i += istride)
	      {
		for (int m = 0; m < xdx; m++)
		  {
		    // TODO: comment about the index access
		    // We store all the columns j such that they have the same j % d
		    // in a contiguous way. The size of such group is dim2 // d.
		    for (int j = m; j < dim2; j += xdx)
		      {
			for (int k = m; k < dim1; k += xdx)
			  {
			    // Same super-block (a super-blocks (b * d, c * d)) ?
			    if ((j / (dim2 / xax)) != (k / (dim1 / xax)))
			      continue;
			    // There is b * c * d non-zero per super-block
			    true_output[MF(i, j, ldc, mf)] +=
			      input[MF(i, k, (bs_last) ? batch_size : lda, (bs_last) ? 1 : mf)] *
			      valuesT[(k / (dim1 / xax)) * xbx * xcx * xdx +
				      m * xbx * xcx +
				      ((k - (dim1 / xax) * (k / (dim1 / xax))) / xdx) * (dim2 / (xax * xdx)) +
				      (j - (dim2 / xax) * (j / (dim2 / xax))) / xdx];
			  }
		      }
		  }
	      }
	  }
      }
    const std::clock_t cpu_end = std::clock();
    printf("cpu clock_t=%f ms\n",
	   1000.0 * (cpu_end - cpu_start) / CLOCKS_PER_SEC);

    // Compute Mean-Square-Error between CPU and kernel results
    mse = 0.0;
    if (hp)
    {
      for (int n = 0; n < nthreads; n++)
      {
        for (int i = n * bpert; i < min(batch_size, (n + 1) * bpert);
             i += istride)
        {
          for (int j = 0; j < dim2; j++)
            mse += pow(__half2float(h_gpu_output[MF(i, j, bs_last ? batch_size : ldc, bs_last ? 1 : mf)]) -
                           true_output[MF(i, j, ldc, mf)],
                       2.0);
        }
      }
    }
    else
    {
      for (int n = 0; n < nthreads; n++)
      {
        for (int i = n * bpert; i < min(batch_size, (n + 1) * bpert);
             i += istride)
        {
          for (int j = 0; j < dim2; j++)
            mse += pow(gpu_output[MF(i, j, bs_last ? batch_size : ldc, bs_last ? 1 : mf)] -
                           true_output[MF(i, j, ldc, mf)],
                       2.0);
        }
      }
    }
    mse = sqrt(mse / ((batch_size / istride) * dim2));
    printf("mse(CPU, GPU)=%e\n", mse);
  }
  else
    mse = 0.0;

  // Write CUDA time (add a threshold ?)
  if ((!hp && mse < 1e-5) || (hp && mse < 1e-1))
  {
    if (meant <= tmin)
    {
      sprintf(name, "%s.out", kernel_name);
      fout = fopen(name, "a");
      if (strcmp(kernel_name, "cublas_factor0_fp16") == 0 ||
          strcmp(kernel_name, "cublas_stride_factor0_fp16") == 0)
        fprintf(fout, "%i %i %i %i %i %.4f %.4f %.4e\n", batch_size, xax, xbx,
                xcx, xdx, meant, stdt,
                mse); //, xCUBLAS_GEMM_ALGOx, xCUBLAS_GEMM_ALGO_TENSOR_OPx);
      else
      {
        if (strcmp(kernel_name, "cublas_factor0_fp32") == 0 ||
            strcmp(kernel_name, "cublas_stride_factor0_fp32") == 0)
          fprintf(fout, "%i %i %i %i %i %.4f %.4f %.4e\n", batch_size, xax,
                  xbx, xcx, xdx, meant, stdt,
                  mse); //, xCUBLAS_GEMM_ALGOx, xCUBLAS_GEMM_ALGO_TENSOR_OPx);
        else
        {
	  fprintf(fout,
		  "%i %i %i %i %i %i %i %i %i %i %i %i %.4f %.4f %.4e\n",
		  batch_size, xax, xbx, xcx, xdx, TILEX, TILEK, TILEY,
		  TX, TY, TILEX, TILEY, meant, stdt, mse);
        }
      }
      fclose(fout);
    }
    else
      printf("Too slow, do not save result.\n");
  }
  else
    {
      printf("Wrong mse, do not save result.\n");
      sprintf(name, "wrong_mse_%s.out", kernel_name);
      fout = fopen(name, "a");
      fprintf(fout, "%i %i %i %i %i %i %i %i %i %i\n",
	      batch_size, xax, xbx, xcx, xdx, TILEX, TILEK, TILEY, TX, TY);
      fclose(fout);
    }

  // ???
  if (check_output)
  {
    int row0 = 0 * (batch_size - 5),
      row1 = row0 + 5 * istride,
      col0 = min(dim2 - 5, 0 * xcx * xdx - 0 * 5);
    if (debug) {
      for (int i = row0; i < row1; i += istride)
	printf("%f %f %f %f %f ...\n", input[MF(i, col0 + 0, bs_last ? batch_size : lda, bs_last ? 1 : mf)],
	       input[MF(i, col0 + 1, bs_last ? batch_size : lda, bs_last ? 1 : mf)],
	       input[MF(i, col0 + 2, bs_last ? batch_size : lda, bs_last ? 1 : mf)],
	       input[MF(i, col0 + 3, bs_last ? batch_size : lda, bs_last ? 1 : mf)],
	       input[MF(i, col0 + 4, bs_last ? batch_size : lda, bs_last ? 1 : mf)]);
    }
    if (0 && debug)
      for (int i = 0; i < NNZ; i++)
        assert(valuesT[i] == 1.0);
    printf("true output:\n");
    for (int i = row0; i < row1; i += istride)
      printf("%f %f %f %f %f ...\n", true_output[MF(i, col0 + 0, ldc, mf)],
             true_output[MF(i, col0 + 1, ldc, mf)],
             true_output[MF(i, col0 + 2, ldc, mf)],
             true_output[MF(i, col0 + 3, ldc, mf)],
             true_output[MF(i, col0 + 4, ldc, mf)]);
    printf("output:\n");
    if (hp)
    {
      if (bs_last)
      {
        for (int i = row0; i < row1; i += istride)
          printf("%f %f %f %f %f ...\n",
                 __half2float(h_gpu_output[MF(i, col0 + 0, batch_size, 1)]),
                 __half2float(h_gpu_output[MF(i, col0 + 1, batch_size, 1)]),
                 __half2float(h_gpu_output[MF(i, col0 + 2, batch_size, 1)]),
                 __half2float(h_gpu_output[MF(i, col0 + 3, batch_size, 1)]),
                 __half2float(h_gpu_output[MF(i, col0 + 4, batch_size, 1)]));
      }
      else
      {
        for (int i = row0; i < row1; i += istride)
          printf("%f %f %f %f %f ...\n",
                 __half2float(h_gpu_output[MF(i, col0 + 0, ldc, mf)]),
                 __half2float(h_gpu_output[MF(i, col0 + 1, ldc, mf)]),
                 __half2float(h_gpu_output[MF(i, col0 + 2, ldc, mf)]),
                 __half2float(h_gpu_output[MF(i, col0 + 3, ldc, mf)]),
                 __half2float(h_gpu_output[MF(i, col0 + 4, ldc, mf)]));
      }
    }
    else
    {
      if (bs_last)
      {
        for (int i = row0; i < row1; i += istride)
          printf("%f %f %f %f %f ...\n",
                 gpu_output[MF(i, col0 + 0, batch_size, 1)],
                 gpu_output[MF(i, col0 + 1, batch_size, 1)],
                 gpu_output[MF(i, col0 + 2, batch_size, 1)],
                 gpu_output[MF(i, col0 + 3, batch_size, 1)],
                 gpu_output[MF(i, col0 + 4, batch_size, 1)]);
      }
      else
      {
        for (int i = row0; i < row1; i += istride)
          printf("%f %f %f %f %f ...\n", gpu_output[MF(i, col0 + 0, ldc, mf)],
                 gpu_output[MF(i, col0 + 1, ldc, mf)],
                 gpu_output[MF(i, col0 + 2, ldc, mf)],
                 gpu_output[MF(i, col0 + 3, ldc, mf)],
                 gpu_output[MF(i, col0 + 4, ldc, mf)]);
      }
    }
  }
  // ???

  // Free memory
  printf("free device memory ...\n");
  if (hp)
  {
    hipFree(d_h_output);
    hipFree(d_h_valuesT);
    hipFree(d_h_input);
    hipFree(d_h_bfactor);
  }
  else
  {
    hipFree(d_output);
    hipFree(d_valuesT);
    hipFree(d_input);
    hipFree(d_bfactor);
  }
  printf("free host memory ...\n\n");
  delete[] input;
  input = NULL;
  delete[] h_input;
  h_input = NULL;
  delete[] valuesT;
  valuesT = NULL;
  delete[] h_valuesT;
  h_valuesT = NULL;
  delete[] bfactor;
  bfactor = NULL;
  delete[] h_bfactor;
  h_bfactor = NULL;
  delete[] gpu_output;
  gpu_output = NULL;
  delete[] h_gpu_output;
  h_gpu_output = NULL;
  delete[] true_output;
  true_output = NULL;

  delete[] ts;
  ts = NULL;

  return 0;
}
